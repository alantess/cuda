
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <math.h>


/* Threads --> Execute Instructions
   Warp --> Comprised on multiple threads -- > Executes the threads in lockstep( Not every threads needs to execute all the instructions )instructions
   Thread Blocks --> Group of threads blocks --> 3 Dimensional (x,y,z) --> assigned to shader core
   Grids --> Composed of thread blocks. --> Figure out how a problem is mapped to gpu --> 3 Dimensional
  
   Blocks sizes are in multiple of 32, so threads must be in the set up in the same way
  */

__global__
void add(int n , float *x, float *y){
  int index = threadIdx.x *blockDim.x + threadIdx.x; // index of the thread within the block
  int stride = blockDim.x * gridDim.x; // Number of threads within the block
  for(int i =index; i<n ; i+= stride){
    y[i] = x[i] + y[i];
  }
}


int main(int argc, char* argv[]){
  int N = 1<<20;
  int blocksize = 256;
  int numBlocks = (N - blocksize-1) / blocksize;
  float *x,*y;
  /* Allocated needed memeory to be accessible from CPU or GPU */ 
  hipMallocManaged(&y, N*sizeof(float));
  hipMallocManaged(&x, N*sizeof(float));
  // Set Values on the host (CPU)
  for(int i = 0; i < N; i++){
    x[i] = 1.0f;
    y[i] = 2.0f;

  }

  // Run kernel on the gpu
  add<<<numBlocks,blocksize>>>(N,x,y);

  // Waits for the GPU to finish before accessing the host
  hipDeviceSynchronize();

  float maxError = 0;
  for(int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] -3.0f));
  std::cout << "Max Error: " << maxError << std::endl;
  
  // Free Memory
  hipFree(x);
  hipFree(y);

  return 0;
}
